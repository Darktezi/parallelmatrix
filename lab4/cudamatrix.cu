#include <iostream>
#include <vector>
#include <string>
#include <chrono>
#include <random>
#include <fstream>
#include <iomanip>
#include "hip/hip_runtime.h"

// CUDA-ядро умножения матриц
__global__ void matMulKernel(const int* A, const int* B, int* C, size_t N) {
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        int sum = 0;
        for (size_t k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << " Error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    for (size_t size = 100; size <= 1000; size += 100) {
        size_t N = size;
        size_t bytes = N * N * sizeof(int);

        // Хостовая память
        std::vector<int> h_A(N * N), h_B(N * N), h_C(N * N);
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_int_distribution<int> dist(1, 10);
        for (size_t i = 0; i < N * N; ++i) {
            h_A[i] = dist(gen);
            h_B[i] = dist(gen);
        }

        // Устройство: выделение памяти
        int *d_A, *d_B, *d_C;
        checkCudaError(hipMalloc(&d_A, bytes), "hipMalloc d_A");
        checkCudaError(hipMalloc(&d_B, bytes), "hipMalloc d_B");
        checkCudaError(hipMalloc(&d_C, bytes), "hipMalloc d_C");

        // Копирование данных на устройство
        checkCudaError(hipMemcpy(d_A, h_A.data(), bytes, hipMemcpyHostToDevice), "Memcpy A");
        checkCudaError(hipMemcpy(d_B, h_B.data(), bytes, hipMemcpyHostToDevice), "Memcpy B");

        // Настройка параметров ядра
        dim3 block(16, 16);
        dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

        // Прогрев ядра
        matMulKernel<<<grid, block>>>(d_A, d_B, d_C, N);
        hipDeviceSynchronize();

        // Измерение времени
        double total_time = 0.0;
        int iterations = 10;
        for (int iter = 0; iter < iterations; ++iter) {
            auto start = std::chrono::high_resolution_clock::now();
            matMulKernel<<<grid, block>>>(d_A, d_B, d_C, N);
            hipDeviceSynchronize();
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> elapsed = end - start;
            total_time += elapsed.count();
        }

        // Копирование результата на хост
        checkCudaError(hipMemcpy(h_C.data(), d_C, bytes, hipMemcpyDeviceToHost), "Memcpy C");

        // Функция записи матрицы в файл
        auto writeMatrix = [&](const std::string& fname, const std::vector<int>& M) {
            std::ofstream fout(fname);
            fout << N << " " << N << "\n";
            for (size_t i = 0; i < N; ++i) {
                for (size_t j = 0; j < N; ++j)
                    fout << M[i * N + j] << " ";
                fout << "\n";
            }
        };

        // Запись файлов
        writeMatrix("matrix_A_" + std::to_string(N) + ".txt", h_A);
        writeMatrix("matrix_B_" + std::to_string(N) + ".txt", h_B);
        writeMatrix("matrix_C_" + std::to_string(N) + ".txt", h_C);

        std::cout << "Size: " << N
                  << " Average CUDA multiplication time: "
                  << (total_time / iterations) << " milliseconds\n";

        // Освобождение памяти устройства
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    return 0;
}
